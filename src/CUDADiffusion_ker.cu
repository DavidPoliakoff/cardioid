//#include "CUDADiffusion.hh"
//#include "DiffusionUtils.hh"
//#include "SymmetricTensor.hh"
//#include <vector>
//#include <map>

//#include "options.h"
//#include "cudautil.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

#define XTILE 20
typedef double Real;


__global__ void diff_6face_v1(const Real* d_psi, Real* d_npsi, const Real* d_sigmaX, const Real* d_sigmaY, const Real* d_sigmaZ,int Lii, int Ljj, int Lkk)
{

  //map z dir to threads
  //z is the fastest varying direction

  //2d decomposition
  //32x32 in y z direction
  __shared__ Real sm_psi[4][32][32]; //32 KB

  #define V0(y,z) sm_psi[pii][y][z]
  #define V1(y,z) sm_psi[cii][y][z]
  #define V2(y,z) sm_psi[nii][y][z]

  #define sigmaX(x,y,z,dir) d_sigmaX[ z + (Lkk-1) * ( y + (Ljj-1) * ( x + (Lii-1) * dir ) ) ]
  #define sigmaY(x,y,z,dir) d_sigmaY[ z + (Lkk-1) * ( y + (Ljj-1) * ( x + (Lii-1) * dir ) ) ]
  #define sigmaZ(x,y,z,dir) d_sigmaZ[ z + (Lkk-1) * ( y + (Ljj-1) * ( x + (Lii-1) * dir ) ) ]

  #define psi(x,y,z) d_psi[ z + Lkk * ( (y) + Ljj * (x) ) ]
  #define npsi(x,y,z) d_npsi[ z + Lkk * ( (y) + Ljj * (x) ) ]

  int tjj = threadIdx.y;
  int tkk = threadIdx.x;

  //shift for each tile
//  d_psi    += 30 * blockIdx.x + Lkk * ( 30 * blockIdx.y );
//  d_npsi   += 30 * blockIdx.x + Lkk * ( 30 * blockIdx.y );
  d_psi    = &(psi(XTILE*blockIdx.x, 30*blockIdx.y, 30*blockIdx.z));
  d_npsi   = &(npsi(XTILE*blockIdx.x, 30*blockIdx.y, 30*blockIdx.z));

  d_sigmaX  = &(sigmaX(XTILE*blockIdx.x-1, 30*blockIdx.y-1, 30*blockIdx.z-1, 0));
  d_sigmaY  = &(sigmaY(XTILE*blockIdx.x-1, 30*blockIdx.y-1, 30*blockIdx.z-1, 0));
  d_sigmaZ  = &(sigmaZ(XTILE*blockIdx.x-1, 30*blockIdx.y-1, 30*blockIdx.z-1, 0));

  int Last_x=XTILE+1; int nLast_y=31; int nLast_z=31;
  if (blockIdx.x == gridDim.x-1) Last_x = Lii-2 - XTILE * blockIdx.x + 1;
  if (blockIdx.y == gridDim.y-1) nLast_y = Ljj-2 - 30 * blockIdx.y + 1;
  if (blockIdx.z == gridDim.z-1) nLast_z = Lkk-2 - 30 * blockIdx.z + 1;

//  if (blockIdx.x==0 && blockIdx.y==0 && blockIdx.z==0) printf("b(%d,%d,%d) t(%d,%d,%d) LastX:%d nLast_y:%d nLast_z:%d %p %p %p %p %p\n",blockIdx.x,blockIdx.y,blockIdx.z,threadIdx.x,threadIdx.y,threadIdx.z,Last_x,nLast_y,nLast_z,&(psi(0,tjj,tkk)),&(npsi(0,tjj,tkk)),&(sigmaX(0,tjj,tkk,0)),&(sigmaY(0,tjj,tkk,0)),&(sigmaZ(0,tjj,tkk,0)));

  if(tjj>nLast_y) return;
  if(tkk>nLast_z) return;

//  d_sigmaX += 30 * blockIdx.x + (Lkk-2) * ( 31 * blockIdx.y );
//  d_sigmaY += 30 * blockIdx.x + (Lkk-2) * ( 31 * blockIdx.y );
//  d_sigmaZ += 31 * blockIdx.x + (Lkk-1) * ( 31 * blockIdx.y );

//  printf("tjj tkk bx by = %d %d %d %d\n",tjj,tkk,blockIdx.x,blockIdx.y);


  int pii,cii,nii,tii;
  pii=0; cii=1; nii=2;

  sm_psi[cii][tkk][tjj] = psi(0,tjj,tkk);
  sm_psi[nii][tkk][tjj] = psi(1,tjj,tkk);
  Real xcharge,ycharge,zcharge,dV;

  __syncthreads();
  //initial
  if ((tkk>0) && (tkk<nLast_z) && (tjj>0) && (tjj<nLast_y))
  {
    Real xd=-V1(tjj,tkk) + V2(tjj,tkk);
    Real yd=(-V1(-1 + tjj,tkk) + V1(1 + tjj,tkk) - V2(-1 + tjj,tkk) + V2(1 + tjj,tkk))/4.;
    Real zd=(-V1(tjj,-1 + tkk) + V1(tjj,1 + tkk) - V2(tjj,-1 + tkk) + V2(tjj,1 + tkk))/4.;

    dV = sigmaX(0,tjj,tkk,0) * xd + sigmaX(0,tjj,tkk,1) * yd + sigmaX(0,tjj,tkk,2) * zd ; 
  }

  tii=pii; pii=cii; cii=nii; nii=tii;

  for(int ii=1;ii<Last_x;ii++)
  {
    sm_psi[nii][tkk][tjj] = psi(ii+1,tjj,tkk);
    __syncthreads();

    // contribution to (ii-1)
    // use link loaded previous
    // y face current
    // tjj=0 calc face at 0-1 and tjj=30 calc face at 30-31
  
    if ((tkk>0) && (tkk<nLast_z) && (tjj<nLast_y))
    {
      Real xd=(-V0(tjj,tkk) - V0(1 + tjj,tkk) + V2(tjj,tkk) + V2(1 + tjj,tkk))/4.;
      Real yd=-V1(tjj,tkk) + V1(1 + tjj,tkk);
      Real zd=(-V1(tjj,-1 + tkk) + V1(tjj,1 + tkk) - V1(1 + tjj,-1 + tkk) + V1(1 + tjj,1 + tkk))/4.;

      ycharge = sigmaY(ii,tjj,tkk,0) * xd + sigmaY(ii,tjj,tkk,1) * yd + sigmaY(ii,tjj,tkk,2) * zd ; 
      dV += ycharge;
      sm_psi[3][tjj][tkk]=ycharge;
    }
    __syncthreads();

    if ((tkk>0) && (tkk<nLast_z) && (tjj>0) && (tjj<nLast_y))
      dV -= sm_psi[3][tjj-1][tkk];  //bring from left

    __syncthreads();

    // z face current
    // tkk=0 calc face at 0-1 and tkk=30 calc face at 30-31
    if ((tkk<nLast_z) && (tjj>0) && (tjj<nLast_y))
    {

      Real xd=(-V0(tjj,tkk) - V0(tjj,1 + tkk) + V2(tjj,tkk) + V2(tjj,1 + tkk))/4.;
      Real yd=(-V1(-1 + tjj,tkk) - V1(-1 + tjj,1 + tkk) + V1(1 + tjj,tkk) + V1(1 + tjj,1 + tkk))/4.;
      Real zd=-V1(tjj,tkk) + V1(tjj,1 + tkk);

      zcharge = sigmaZ(ii,tjj,tkk,0) * xd + sigmaZ(ii,tjj,tkk,1) * yd + sigmaZ(ii,tjj,tkk,2) * zd ; 
      dV += zcharge;
      sm_psi[3][tjj][tkk]=zcharge;
    }

    __syncthreads();

    if ((tkk>0) && (tkk<nLast_z) && (tjj>0) && (tjj<nLast_y))
      dV -= sm_psi[3][tjj][tkk-1];

    //__syncthreads();

    // x face current
    if ((tkk>0) && (tkk<nLast_z) && (tjj>0) && (tjj<nLast_y))
    {
      Real xd=-V1(tjj,tkk) + V2(tjj,tkk);
      Real yd=(-V1(-1 + tjj,tkk) + V1(1 + tjj,tkk) - V2(-1 + tjj,tkk) + V2(1 + tjj,tkk))/4.;
      Real zd=(-V1(tjj,-1 + tkk) + V1(tjj,1 + tkk) - V2(tjj,-1 + tkk) + V2(tjj,1 + tkk))/4.;
 
      xcharge = sigmaX(ii,tjj,tkk,0) * xd + sigmaX(ii,tjj,tkk,1) * yd + sigmaX(ii,tjj,tkk,2) * zd ; 
      dV += xcharge;
      //store dV
      npsi(ii,tjj,tkk) = dV;

      dV = -xcharge; //pass to the next cell in x-dir
    }
    tii=pii; pii=cii; cii=nii; nii=tii;
  }
//  #undef V0(y,z)
//  #undef V1(y,z)
//  #undef V2(y,z)
//  #undef sigmaX(x,y,z,dir) 
//  #undef sigmaY(x,y,z,dir) 
//  #undef sigmaZ(x,y,z,dir) 
//  #undef psi(x,y,z) 
//  #undef npsi(x,y,z) 
}

__global__ void map_dVm(double * dVmT, double* dVm, const int *remap,int nCells)
{
  int idx0 = threadIdx.x + blockDim.x*blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int idx = idx0 ; idx<nCells ; idx+=stride)
      dVmT[idx] = dVm[remap[idx]];
}

//__global__ void map_V(double * VT, double* V, const int *remap,int nCells)
//{
//  int idx0 = threadIdx.x + blockDim.x*blockIdx.x;
//  int stride = blockDim.x * gridDim.x;
//  for(int idx = idx0 ; idx<nCells ; idx+=stride)
//      VT[remap[idx]] = V[idx];
//}

extern "C"
{
void call_cuda_kernels(const Real *VmRaw, Real *dVmRaw, const Real *sigmaRaw, int nx, int ny, int nz, Real *dVmOut, const int *lookup,int nCells)
{
   //determine block dim
   //1. blockdim.z and blockdim.y are determined in a simple way.
   int bdimz = (int)((nz-2)/30) + ((nz-2)%30==0?0:1);
   int bdimy = (int)((ny-2)/30) + ((ny-2)%30==0?0:1);
   int bdimx = (int)((nx-2)/XTILE) + ((nx-2)%XTILE==0?0:1);
   
//   printf("Vm=%p dVm=%p sigma=%p \n",VmRaw,dVmRaw,sigmaRaw);
//   printf("call_cuda_kernels %d,%d,%d (%d,%d,%d)\n",nx,ny,nz,bdimx,bdimy,bdimz);
#ifdef GPU_SM_70
   cudaFuncSetAttribute(diff_6face_v1, cudaFuncAttributePreferredSharedMemoryCarveout, 50);
#endif

   //map_V<<<112,512>>>(VmBlockRaw,VmRaw,lookup,nCells);
   diff_6face_v1<<<dim3(bdimx,bdimy,bdimz),dim3(32,32,1)>>>(VmRaw,dVmRaw,sigmaRaw,sigmaRaw+3*(nx-1)*(ny-1)*(nz-1),sigmaRaw+6*(nx-1)*(ny-1)*(nz-1),nx,ny,nz);
   map_dVm<<<112,512>>>(dVmRaw,dVmOut,lookup,nCells);
}
}
