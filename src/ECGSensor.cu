#include "hip/hip_runtime.h"
#include "Ledger.hh"
#include "TransportCoordinator.hh"
#include "Long64.hh"
#include <stdio.h>

#define NUM_TBLOCK 112
#define CELL_PER_BLOCK 32


__global__ void calcInvr_kernel(ArrayView<double> invr,
                                ConstArrayView<Long64> gids,
                                ConstArrayView<double> ecgPoints,
                                const int nEcgPoints,
                                const int nx, const int ny, const int nz,
                                const double dx, const double dy, const double dz,
                                const int begin, const int end)
{

   int ii = threadIdx.x + blockIdx.x*blockDim.x  +begin;
   if (ii >= end) { return; }

    const int dim=3;

    Long64 gid=gids[ii];
    int x=gid%nx;
    int y=(gid/nx) %ny;
    int z=gid/nx/ny;

    double xcoor=x*dx;
    double ycoor=y*dy;
    double zcoor=z*dz;

    for(int j=0; j<nEcgPoints; ++j)
    {
        double dxx=xcoor-ecgPoints[j*dim];
        double dyy=ycoor-ecgPoints[j*dim+1];
        double dzz=zcoor-ecgPoints[j*dim+2];
        invr[ii*nEcgPoints+j]=1.0/sqrt(dxx*dxx+dyy*dyy+dzz*dzz);
    }

}

void calcInvrCUDA(OnDevice<ArrayView<double>> invr,
                  OnDevice<ConstArrayView<Long64>> gids, 
                  OnDevice<ConstArrayView<double>> ecgPoints,
                  const int nEcgPoints,
                  const int nx, const int ny, const int nz,
                  const double dx, const double dy, const double dz)
{
    int blockSize = 1024;
    ConstArrayView<Long64> tmp=gids;

    int begin=0;
    int end=tmp.size();


    calcInvr_kernel<<<(end-begin+blockSize-1)/blockSize, blockSize>>>
        (invr,
         gids,
         ecgPoints,
         nEcgPoints,
         nx, ny, nz,
         dx, dy, dz,
         begin, end);

}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
inline __device__ double atomicAdd(double* pointer, const double value)
{
   unsigned long long int* address_as_ull = reinterpret_cast<unsigned long long int*>(pointer);
   unsigned long long int old = *address_as_ull;
   unsigned long long int assumed;
   do {
      assumed=old;
      old = atomicCAS(address_as_ull,
                      assumed,
                      __double_as_longlong(value + __longlong_as_double(assumed)));
   } while(assumed != old);
   return assumed;
}
#endif

__global__ void calcEcg_kernel(ArrayView<double> ecgs,
                               ConstArrayView<double> invr,
                               ConstArrayView<double> dVmDiffusion,
                               const int nEcgPoints,
                               const int cellPartition,
                               const int nCells)
{
  __shared__ double smResult[1024];
  double tmpECG=0;

  const int cellStart = threadIdx.x / nEcgPoints + blockIdx.x * cellPartition;
  const int cellStride = blockDim.x / nEcgPoints;
  const int cellEnd = (blockIdx.x<(NUM_TBLOCK-1))?((blockIdx.x+1)*cellPartition):(nCells);
  const int ecgID = threadIdx.x % nEcgPoints;

  const unsigned int ecgSet = blockDim.x/ nEcgPoints;    

  if ( threadIdx.x >= ecgSet * nEcgPoints ) return;

  for(int cell=cellStart;cell<cellEnd;cell += cellStride)
  {
    tmpECG +=dVmDiffusion[cell] * invr[ecgID + nEcgPoints*cell];
  }

  const unsigned int close2N = 0x80000000 >> __clz(ecgSet-1);

  if ( threadIdx.x >= close2N*nEcgPoints )
      smResult[threadIdx.x] = tmpECG;
  __syncthreads();
  if (threadIdx.x < (ecgSet - close2N)*nEcgPoints )
      tmpECG += smResult[threadIdx.x + close2N*nEcgPoints];

  int jump = close2N >> 1;
  while(jump>0)
  {
    if (( threadIdx.x >= jump*nEcgPoints ) && ( threadIdx.x < 2*jump*nEcgPoints ))
      smResult[threadIdx.x] = tmpECG;
    __syncthreads();
    if(threadIdx.x < jump*nEcgPoints)
      tmpECG += smResult[threadIdx.x + jump*nEcgPoints];
    jump = jump >> 1;
  }

  if(threadIdx.x<nEcgPoints){
    double* result=&ecgs[0];
    atomicAdd(result+ecgID ,tmpECG);
   }
   
}

void calcEcgCUDA(OnDevice<ArrayView<double>> ecgs,
                 OnDevice<ConstArrayView<double>> invr,
                 OnDevice<ConstArrayView<double>> dVmDiffusion,
                 const int nEcgPoints)
{

    ConstArrayView<double> tmp=dVmDiffusion;
    int nCells=tmp.size();
    const int cellPartition = (nCells+(NUM_TBLOCK-1))/(NUM_TBLOCK);

    if ( nEcgPoints > 32 ) printf("error:too many ECG points\n");
    calcEcg_kernel<<<NUM_TBLOCK, (nEcgPoints*CELL_PER_BLOCK)>>>
        (ecgs,
         invr,
         dVmDiffusion,
         nEcgPoints,
         cellPartition,
         nCells);
}

__global__ void dump_kernel(ConstArrayView<double> data)
{
	for(int ii=0; ii<100; ii++)
        {
	    printf("%f ",data[ii]);
	    if((ii+1)%10==0) printf("\n");
        }
}

void dump_GPU_data(OnDevice<ArrayView<double>> ecgs,
                 OnDevice<ConstArrayView<double>> invr,
                 OnDevice<ConstArrayView<double>> dVmDiffusion,
                 const int nEcgPoints)
{

    dump_kernel<<<1,1>>>(invr);
    printf("\n\n\n");
    dump_kernel<<<1,1>>>(dVmDiffusion);
    printf("\n\n\n");
}
