#include "hip/hip_runtime.h"
#include "Ledger.hh"
#include "TransportCoordinator.hh"
#include "Long64.hh"

#define NUM_TBLOCK 160
#define CELL_PER_BLOCK 32


__global__ void calcInvr_kernel(ArrayView<double> invr,
                                ConstArrayView<Long64> gids,
                                ConstArrayView<double> ecgPoints,
                                const int nEcgPoints,
                                const int nx, const int ny, const int nz,
                                const double dx, const double dy, const double dz,
                                const int begin, const int end)
{

   int ii = threadIdx.x + blockIdx.x*blockDim.x  +begin;
   if (ii >= end) { return; }

    const int dim=3;

    Long64 gid=gids[ii];
    int x=gid%nx;
    int y=gid/nx%ny;
    int z=gid/nx/ny;

    double xcoor=x*dx;
    double ycoor=y*dy;
    double zcoor=z*dz;

    for(int j=0; j<nEcgPoints; ++j)
    {
        double dxx=xcoor-ecgPoints[j*dim];
        double dyy=ycoor-ecgPoints[j*dim+1];
        double dzz=zcoor-ecgPoints[j*dim+2];
        invr[ii*nEcgPoints+j]=1.0/sqrt(dxx*dxx+dyy*dyy+dzz*dzz);
    }

}

void calcInvrCUDA(OnDevice<ArrayView<double>> invr,
                  OnDevice<ConstArrayView<Long64>> gids, 
                  OnDevice<ConstArrayView<double>> ecgPoints,
                  const int nEcgPoints,
                  const int nx, const int ny, const int nz,
                  const double dx, const double dy, const double dz)
{
    int blockSize = 1024;
    ConstArrayView<Long64> tmp=gids;

    int begin=0;
    int end=tmp.size();


    calcInvr_kernel<<<(end-begin+blockSize-1)/blockSize, blockSize>>>
        (invr,
         gids,
         ecgPoints,
         nEcgPoints,
         nx, ny, nz,
         dx, dy, dz,
         begin, end);

}

__global__ void calcEcg_kernel(ArrayView<double> ecgs,
                               ConstArrayView<double> invr,
                               ConstArrayView<double> Vm,
                               const int nEcgPoints,
                               const int cellPartition,
                               const int nCells)
{
  __shared__ double smResult[1024];
  double tmpECG=0;

  const int cellStart = threadIdx.x / nEcgPoints + blockIdx.x * cellPartition;
  const int cellStride = blockDim.x / nEcgPoints;
  const int cellEnd = (blockIdx.x<(NUM_TBLOCK-1))?((blockIdx.x+1)*cellPartition):(nCells);
  const int ecgID = threadIdx.x % nEcgPoints;
  const unsigned int ecgSet = 1024 / nEcgPoints;    

  if ( threadIdx.x >= ecgSet * nEcgPoints ) return;

  for(int cell=cellStart;cell<cellEnd;cell += cellStride)
  {
    tmpECG +=Vm[cell] * invr[ecgID + nEcgPoints*cell];
  }

  const unsigned int close2N = 0x80000000 >> __clz(ecgSet-1);

  if ( threadIdx.x >= close2N*nEcgPoints )
      smResult[threadIdx.x] = tmpECG;
  __syncthreads();
  if (threadIdx.x < (ecgSet - close2N)*nEcgPoints )
      tmpECG += smResult[threadIdx.x + close2N*nEcgPoints];

  int jump = close2N >> 1;
  while(jump>0)
  {
    if (( threadIdx.x >= jump*nEcgPoints ) && ( threadIdx.x < 2*jump*nEcgPoints ))
      smResult[threadIdx.x] = tmpECG;
    __syncthreads();
    if(threadIdx.x < jump*nEcgPoints)
      tmpECG += smResult[threadIdx.x + jump*nEcgPoints];
    jump = jump >> 1;
  }

  if(threadIdx.x<nEcgPoints){
    double* result=&ecgs[0];
    atomicAdd(result+ecgID ,tmpECG);
   }
   
}

void calcEcgCUDA(OnDevice<ArrayView<double>> ecgs,
                 OnDevice<ConstArrayView<double>> invr,
                 OnDevice<ConstArrayView<double>> Vm,
                 const int nEcgPoints)
{

    ConstArrayView<double> tmp=Vm;
    int nCells=tmp.size();
    const int cellPartition = (nCells+(NUM_TBLOCK-1))/(NUM_TBLOCK);

    calcEcg_kernel<<<NUM_TBLOCK, (nEcgPoints*CELL_PER_BLOCK)>>>
        (ecgs,
         invr,
         Vm,
         nEcgPoints,
         cellPartition,
         nCells);
}
